#include "hip/hip_runtime.h"
#include "reconstr.h"
#include <hip/hip_runtime.h>


__device__
Real sign(Real x) {
    if (x < 0.) {
        return -1.;
    } else if (x > 0.) {
        return 1.;
    } else {
        return 0.;
    }
}


__device__
Real minmod(Real x, Real y) {
    if (sign(x) != sign(y)) return 0.;
    return sign(x) * (fabs(x) < fabs(y) ? fabs(x) : fabs(y));
}


__device__
void CONST(Real *u, Real &ul, Real &ur) {
    ul = u[K_WENO - 1];
    ur = u[K_WENO];
}


__device__
void TVD2(Real *u, Real &ul, Real &ur) {
    ul = u[K_WENO - 1] + 0.5 * minmod(u[K_WENO - 1] - u[K_WENO - 2], u[3] - u[K_WENO - 1]);
    ur = u[3] - 0.5 * minmod(u[K_WENO] - u[K_WENO - 1], u[K_WENO + 1] - u[K_WENO]);
}


__device__
void WENO5(Real *u, Real &ul, Real &ur) {
    Real beta[3];
    Real alpha[3];
    Real eps = 1.0e-6;
    if ((u[2] - u[1]) * (u[3] - u[2]) < 0.0) ul = u[2];
    else {
        //значение слева
        beta[0] = (13. / 12.) * (u[2] - 2 * u[3] + u[4]) * (u[2] - 2 * u[3] + u[4]) +
                  0.25 * (3 * u[2] - 4 * u[3] + u[4]) * (3 * u[2] - 4 * u[3] + u[4]);
        beta[1] = (13. / 12.) * (u[1] - 2 * u[2] + u[3]) * (u[1] - 2 * u[2] + u[3]) +
                  0.25 * (u[1] - u[3]) * (u[1] - u[3]);
        beta[2] = (13. / 12.) * (u[0] - 2 * u[1] + u[2]) * (u[0] - 2 * u[1] + u[2]) +
                  0.25 * (u[0] - 4 * u[1] + 3 * u[2]) * (u[0] - 4 * u[1] + 3 * u[2]);
        alpha[0] = 0.3 / ((eps + beta[0]) * (eps + beta[0]));
        alpha[1] = 0.6 / ((eps + beta[1]) * (eps + beta[1]));
        alpha[2] = 0.1 / ((eps + beta[2]) * (eps + beta[2]));
        ul = (alpha[0] * (2 * u[2] + 5 * u[3] - u[4]) + alpha[1] * (-u[1] + 5 * u[2] + 2 * u[3]) +
              alpha[2] * (2 * u[0] - 7 * u[1] + 11 * u[2])) / ((alpha[0] + alpha[1] + alpha[2]) * 6);
    }
    if ((u[3] - u[2]) * (u[4] - u[3]) < 0.0) ur = u[3];
    else {
        //значение справа
        beta[0] = (13. / 12.) * (u[3] - 2 * u[4] + u[5]) * (u[3] - 2 * u[4] + u[5]) +
                  0.25 * (3 * u[3] - 4 * u[4] + u[5]) * (3 * u[3] - 4 * u[4] + u[5]);
        beta[1] = (13. / 12.) * (u[2] - 2 * u[3] + u[4]) * (u[2] - 2 * u[3] + u[4]) +
                  0.25 * (u[2] - u[4]) * (u[2] - u[4]);
        beta[2] = (13. / 12.) * (u[1] - 2 * u[2] + u[3]) * (u[1] - 2 * u[2] + u[3]) +
                  0.25 * (u[1] - 4 * u[2] + 3 * u[3]) * (u[1] - 4 * u[2] + 3 * u[3]);
        alpha[0] = 0.1 / ((eps + beta[0]) * (eps + beta[0]));
        alpha[1] = 0.6 / ((eps + beta[1]) * (eps + beta[1]));
        alpha[2] = 0.3 / ((eps + beta[2]) * (eps + beta[2]));
        ur = (alpha[0] * (11 * u[3] - 7 * u[4] + 2 * u[5]) + alpha[1] * (2 * u[2] + 5 * u[3] - u[4]) +
              alpha[2] * (-u[1] + 5 * u[2] + 2 * u[3])) / ((alpha[0] + alpha[1] + alpha[2]) * 6);
    }
}
